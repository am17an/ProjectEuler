#include <iostream>
#include <vector> // Using vector for easier host-side matrix handling
#include <hip/hip_runtime.h>
#include <stdexcept> // For error handling

// Define constants based on the problem
const int N = 2000;
const int32_t MOD = 20092010;
const int64_t K = 1000000000000000000LL; // k = 10^18

// --- CUDA Error Handling Macro ---
#define CUDA_CHECK(call)                                                  \
do {                                                                      \
    hipError_t err = call;                                               \
    if (err != hipSuccess) {                                             \
        fprintf(stderr, "CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(err));                                 \
        throw std::runtime_error(hipGetErrorString(err));                \
    }                                                                     \
} while (0)

// --- GPU Matrix Multiplication Kernel (Unchanged) ---
__global__ void matmul_mod(const int64_t* A, const int64_t* B, int64_t* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int64_t val = 0;
        for (int k = 0; k < N; ++k) {
            // Fetch a and b, ensuring they are already mod MOD if matrices A, B contain results from previous steps
            // Note: If initial A, B elements could be >> MOD or negative, applying initial modulo is crucial.
            // Here, intermediate results are always kept modulo MOD.
            int64_t a_val = A[row * N + k];
            int64_t b_val = B[k * N + col];

            // Intermediate product (a_val * b_val) must fit in int64_t.
            // max(a_val*b_val) ~ (MOD-1)*(MOD-1) ~ 4e14, which fits in int64_t (max ~9e18)
            int64_t product = (a_val * b_val) % MOD;

            // Accumulate value
            val = (val + product) % MOD;
        }
        // Ensure result is positive
        C[row * N + col] = (val + MOD) % MOD;
    }
}

// --- GPU Matrix Multiplication Function (Error handling added) ---
void gpu_matrix_multiply(int64_t* d_A, int64_t* d_B, int64_t* d_C) {
    dim3 threadsPerBlock(16, 16);
    // Calculate blocks needed, ensuring coverage
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmul_mod<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    // Check for kernel launch errors immediately
    CUDA_CHECK(hipGetLastError());
    // Synchronize device to ensure computation is complete and check errors
    CUDA_CHECK(hipDeviceSynchronize());
}

// --- GPU Matrix Exponentiation (Corrected Initialization) ---
// Computes (h_matrix ^ exp) mod MOD and stores result back in h_matrix
void matrix_expo_gpu(int64_t* h_matrix, int64_t exp) {
    if (exp == 0) {
        // If exp is 0, result is Identity matrix
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                h_matrix[i * N + j] = (i == j) ? 1 : 0;
            }
        }
        return;
    }
    if (exp == 1) {
        // If exp is 1, result is the matrix itself (modulo MOD)
        // Ensure initial matrix is mod MOD if necessary (depends on how it was created)
         for(size_t i = 0; i < (size_t)N * N; ++i) {
             h_matrix[i] = (h_matrix[i] % MOD + MOD) % MOD;
         }
        return;
    }


    int64_t *d_matrix = nullptr, *d_temp = nullptr, *d_result = nullptr, *d_identity = nullptr;
    size_t matrix_size_bytes = sizeof(int64_t) * N * N;

    try {
        // 1. Prepare Identity Matrix on Device
        std::vector<int64_t> h_identity(N * N, 0);
        for (int i = 0; i < N; ++i) {
            h_identity[i * N + i] = 1;
        }
        CUDA_CHECK(hipMalloc(&d_identity, matrix_size_bytes));
        CUDA_CHECK(hipMemcpy(d_identity, h_identity.data(), matrix_size_bytes, hipMemcpyHostToDevice));
        // Host vector h_identity goes out of scope or clear it if needed std::vector<int64_t>().swap(h_identity);

        // 2. Allocate device memory for matrices
        CUDA_CHECK(hipMalloc(&d_matrix, matrix_size_bytes));
        CUDA_CHECK(hipMalloc(&d_temp, matrix_size_bytes));
        CUDA_CHECK(hipMalloc(&d_result, matrix_size_bytes));

        // 3. Copy input matrix (base) to d_matrix
        CUDA_CHECK(hipMemcpy(d_matrix, h_matrix, matrix_size_bytes, hipMemcpyHostToDevice));
        // Ensure base matrix elements are mod MOD
        // This could be a kernel or done during copy, for simplicity we assume gpu_matrix_multiply handles it

        // 4. Initialize result matrix (d_result) to Identity
        CUDA_CHECK(hipMemcpy(d_result, d_identity, matrix_size_bytes, hipMemcpyDeviceToDevice));

        // 5. Perform exponentiation by squaring
        while (exp > 0) {
             std::cout << "Exponent remaining: " << exp << std::endl; // Progress indicator
            if (exp % 2 == 1) {
                // result = result * matrix_power (d_result = d_result * d_matrix)
                gpu_matrix_multiply(d_result, d_matrix, d_temp);
                std::swap(d_result, d_temp); // d_result now holds the product
            }
            // matrix_power = matrix_power * matrix_power (d_matrix = d_matrix * d_matrix)
            gpu_matrix_multiply(d_matrix, d_matrix, d_temp);
            std::swap(d_matrix, d_temp); // d_matrix now holds the square
            exp /= 2;
        }

        // 6. Copy final result back from d_result to host h_matrix
        CUDA_CHECK(hipMemcpy(h_matrix, d_result, matrix_size_bytes, hipMemcpyDeviceToHost));

    } catch (const std::exception& e) {
        std::cerr << "An error occurred: " << e.what() << std::endl;
        // Cleanup allocated memory even if error occurred
        if (d_matrix) hipFree(d_matrix);
        if (d_temp) hipFree(d_temp);
        if (d_result) hipFree(d_result);
        if (d_identity) hipFree(d_identity);
        throw; // Re-throw exception
    }

    // 7. Free all allocated device memory
    CUDA_CHECK(hipFree(d_matrix));
    CUDA_CHECK(hipFree(d_temp));
    CUDA_CHECK(hipFree(d_result));
    CUDA_CHECK(hipFree(d_identity));
}


int main() {
    // Use std::vector for easier host matrix management
    std::vector<int64_t> h_matrix(N * N, 0);

    // --- Initialize the CORRECT Transition Matrix T ---
    // T[0][1998] = 1, T[0][1999] = 1
    h_matrix[0 * N + 1998] = 1;
    h_matrix[0 * N + 1999] = 1;
    // T[i][i-1] = 1 for i = 1 to N-1
    for (int i = 1; i < N; ++i) {
        h_matrix[i * N + (i - 1)] = 1;
    }

    // --- Set the CORRECT Exponent ---
    // We need T^(k - (N-1)) = T^(k - 1999)
    int64_t exponent = K - (N - 1); // k - 1999
     std::cout << "Target Exponent: " << exponent << std::endl;


    // --- Perform Matrix Exponentiation ---
    try {
        matrix_expo_gpu(h_matrix.data(), exponent); // Pass pointer to vector data

        // --- Calculate the final answer g_k ---
        // g_k = sum of the first row of the resulting matrix M = T^p
        int64_t ans = 0;
        for (int i = 0; i < N; ++i) {
            ans = (ans + h_matrix[0 * N + i]) % MOD;
        }
        // Ensure final answer is positive
        ans = (ans + MOD) % MOD;

        std::cout << "Calculated g_k mod M = " << ans << std::endl;

    } catch (const std::exception& e) {
        std::cerr << "Matrix exponentiation failed." << std::endl;
        return 1; // Indicate failure
    }

    return 0; // Indicate success
}